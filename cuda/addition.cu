#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1

__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x;
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}

int main(void) {

    int *a, *b, *c; // Ponteiros para a memória do host
    int *d_a, *d_b, *d_c; // Ponteiros para a memória do dispositivo
    int size = N * sizeof(int);

    // Alocação de memória no host
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    // Inicializando os arrays no host
    for (int i = 0; i < N; i++) {
        a[i] = 2; // Exemplo de valor
        b[i] = 7; // Exemplo de valor
    }

    // Alocação de memória no dispositivo
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Cópia dos dados do host para o dispositivo
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Chamada do kernel com N threads
    add<<<1, N>>>(d_a, d_b, d_c);

    // Cópia dos resultados do dispositivo para o host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Exibir os resultados
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Liberação da memória
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
